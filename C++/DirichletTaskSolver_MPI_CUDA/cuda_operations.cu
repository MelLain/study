#include "hip/hip_runtime.h"
#include <vector>

#include "mpi_helpers.h"
#include "cuda_operations.h"

namespace DTS {

void Clear_CUDA(double* src) {
  CheckCuda(hipFree(src));
}

void PrintMatrix_CUDA(double* src, size_t num_rows, size_t num_cols) {
  size_t num_elems = num_rows * num_cols;
  double* dst = new double[num_elems];
  CheckCuda(hipMemcpy(dst, src, num_elems * sizeof(double), hipMemcpyDeviceToHost));
  for (size_t i = 0; i < num_rows; ++i) {
    for (size_t j = 0; j < num_cols; ++j) {
      std::cout << dst[i * num_cols + j] << "  ";
    }
    std::cout << std::endl;
  }
  delete[] dst;
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void CopyMatrix_CUDA(double* dst, double* src, size_t num_rows, size_t num_cols) {
  hipDeviceProp_t devProp;
  CheckCuda(hipGetDeviceProperties(&devProp, 0));

  int num_elems = num_rows * num_cols;
  int temp = (int)(devProp.maxThreadsPerBlock / 2);
  int num_threads = (temp > num_elems) ? num_elems : temp;
  int num_blocks = (num_elems - 1) / num_threads + 1;
  CopyMatrixImpl<<<num_blocks, num_threads>>>(dst, src, num_elems);
}

__global__ void CopyMatrixImpl(double* dst, double* src, size_t num_elems) {
  int i = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (i < num_elems) {
    dst[i] = src[i];
  }
}

void SumMatrices_CUDA(double* dst, double* src_1, double* src_2, size_t num_rows, size_t num_cols) {
  hipDeviceProp_t devProp;
  CheckCuda(hipGetDeviceProperties(&devProp, 0));

  int num_elems = num_rows * num_cols;
  int temp = (int)(devProp.maxThreadsPerBlock / 2);
  int num_threads = (temp > num_elems) ? num_elems : temp;
  int num_blocks = (num_elems - 1) / num_threads + 1;
  SumMatricesImpl<<<num_blocks, num_threads>>>(dst, src_1, src_2, num_elems);
}

__global__ void SumMatricesImpl(double* dst, double* src_1, double* src_2, size_t num_elems) {
  int i = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (i < num_elems) {
    dst[i] = src_1[i] + src_2[i];
  }
}

void DiffMatrices_CUDA(double* dst, double* src_1, double* src_2, size_t num_rows, size_t num_cols) {
  hipDeviceProp_t devProp;
  CheckCuda(hipGetDeviceProperties(&devProp, 0));

  int num_elems = num_rows * num_cols;
  int temp = (int)(devProp.maxThreadsPerBlock / 2);
  int num_threads = (temp > num_elems) ? num_elems : temp;
  int num_blocks = (num_elems - 1) / num_threads + 1;
  DiffMatricesImpl<<<num_blocks, num_threads>>>(dst, src_1, src_2, num_elems);
}

__global__ void DiffMatricesImpl(double* dst, double* src_1, double* src_2, size_t num_elems) {
  int i = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (i < num_elems) {
    dst[i] = src_1[i] - src_2[i];
  }
}

void ProdMatrixByScalar_CUDA(double* dst, double* src, double alpha, size_t num_rows, size_t num_cols) {
  hipDeviceProp_t devProp;
  CheckCuda(hipGetDeviceProperties(&devProp, 0));

  int num_elems = num_rows * num_cols;
  int temp = (int)(devProp.maxThreadsPerBlock / 2);
  int num_threads = (temp > num_elems) ? num_elems : temp;
  int num_blocks = (num_elems - 1) / num_threads + 1;
  ProdMatrixByScalarImpl<<<num_blocks, num_threads>>>(dst, src, alpha, num_elems);
}

__global__ void ProdMatrixByScalarImpl(double* dst, double* src, double alpha, size_t num_elems) {
  int i = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (i < num_elems) {
    dst[i] = src[i] * alpha;
  }
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void FivePointsLaplass_CUDA(double* dst, double* src, double* grid_x, double* grid_y, size_t num_rows, size_t num_cols) {
  hipDeviceProp_t devProp;
  CheckCuda(hipGetDeviceProperties(&devProp, 0));

  int temp = (int)(sqrt(devProp.maxThreadsPerBlock) / 2);
  int num_threads_x = (temp > num_rows) ? num_rows : temp;
  int num_threads_y = (temp > num_cols) ? num_cols : temp;
  dim3 num_threads(num_threads_x, num_threads_y);
  dim3 num_blocks((num_rows - 1) / num_threads_x + 1, (num_cols - 1) / num_threads_y + 1);

  FivePointsLaplassImpl<<<num_blocks, num_threads>>>(dst, src, grid_x, grid_y, num_rows, num_cols);
}

__global__ void FivePointsLaplassImpl(double* dst, double* src, double* grid_x, double* grid_y, size_t num_rows, size_t num_cols) {
  int i = (blockIdx.x * blockDim.x) + threadIdx.x;
  int j = (blockIdx.y * blockDim.y) + threadIdx.y;

  if (i < (num_rows - 1) && j < (num_cols - 1) && i > 0 && j > 0) {
    double part_1 = (matrix_get(src, i, j, num_cols) - matrix_get(src, i - 1, j, num_cols)) / grid_r_step(grid_x, grid_y, i, num_cols) -
      (matrix_get(src, i + 1, j, num_cols) - matrix_get(src, i, j, num_cols)) / grid_r_step(grid_x, grid_y, i + 1, num_cols);
    double part_2 = (matrix_get(src, i, j, num_cols) - matrix_get(src, i, j - 1, num_cols)) / grid_c_step(grid_x, grid_y, j, num_cols) -
      (matrix_get(src, i, j + 1, num_cols) - matrix_get(src, i, j, num_cols)) / grid_c_step(grid_x, grid_y, j + 1, num_cols);
    matrix_set(dst, i, j, 2 * part_1 / (grid_r_step(grid_x, grid_y, i, num_cols) + grid_r_step(grid_x, grid_y, i + 1, num_cols)) +
                          2 * part_2 / (grid_c_step(grid_x, grid_y, j, num_cols) + grid_c_step(grid_x, grid_y, j + 1, num_cols)), num_cols);
  }

  if (((i == 0 || i == (num_rows - 1)) && (j < num_cols)) || ((j == 0 || j == (num_cols - 1)) && (i < num_rows))) {
    matrix_set(dst, i, j, 0.0, num_cols);
  }
}

double ProductByPointAndSum_CUDA(double* src_1, double* src_2, double* grid_x, double* grid_y, size_t num_rows, size_t num_cols) {
  // http://mc.stanford.edu/cgi-bin/images/5/55/Darve_cme343_cuda_4.pdf
  hipDeviceProp_t devProp;
  CheckCuda(hipGetDeviceProperties(&devProp, 0));

  double* pre_retval;
  double* pre_retval_cuda;

  int num_elems = num_cols * num_rows;
  int temp = (int)(devProp.maxThreadsPerBlock / 2);
  int num_threads = (temp > num_elems) ? num_elems : temp;
  int num_blocks = (num_elems - 1) / num_threads + 1;

  pre_retval = new double[num_blocks + 1];
  CheckCuda(hipMalloc((void**)(&pre_retval_cuda), (num_blocks + 1) * sizeof(double)));

  ProductByPointAndSumImpl<<<num_blocks, num_threads, num_threads * sizeof(double)>>>(pre_retval_cuda, src_1, src_2,
                                                                                      grid_x, grid_y, num_rows, num_cols);

  CheckCuda(hipMemcpy(pre_retval, pre_retval_cuda, num_blocks * sizeof(double), hipMemcpyDeviceToHost));

  double retval = 0.0;
  for (size_t i = 0; i < num_blocks; ++i) {
    retval += pre_retval[i];
  }
  return retval;
}

__global__ void ProductByPointAndSumImpl(double* pre_retval, double* src_1, double* src_2,
                                         double* grid_x, double* grid_y, size_t num_rows, size_t num_cols) {
  extern __shared__ double shared_mem[];
  int tid = threadIdx.x;
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  int num_elems = num_rows * num_cols;
  int col = (i % num_cols);
  int row = (i / num_cols);
  shared_mem[tid] = ((i > num_cols) && (i < (num_elems - num_cols)) && (col > 0) && (col < (num_cols - 1))) ?
                    src_1[i] * src_2[i] * 0.25 *
    (grid_r_step(grid_x, grid_y, row, num_cols) + grid_r_step(grid_x, grid_y, row + 1, num_cols)) *
    (grid_c_step(grid_x, grid_y, col, num_cols) + grid_c_step(grid_x, grid_y, col + 1, num_cols)) : 0.0;
  __syncthreads();

  for (int k = blockDim.x / 2; k > 0; k /= 2) {
    if (tid < k) {
      shared_mem[tid] += shared_mem[tid + k];
    }
    __syncthreads();
  }

  if (tid == 0) {
    pre_retval[blockIdx.x] = shared_mem[0];
  }
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void InitValues_CUDA(double** values, double* grid_x, double* grid_y,
                     size_t num_rows, size_t num_cols, const ProcBounds& proc_bounds) {
  int num_elems = num_rows * num_cols;
  CheckCuda(hipMalloc(values, num_elems * sizeof(double)));

  hipDeviceProp_t devProp;
  CheckCuda(hipGetDeviceProperties(&devProp, 0));

  int temp = (int)(sqrt(devProp.maxThreadsPerBlock) / 2);
  int num_threads_x = (temp > num_rows) ? num_rows : temp;
  int num_threads_y = (temp > num_cols) ? num_cols : temp;
  dim3 num_threads(num_threads_x, num_threads_y);
  dim3 num_blocks((num_rows - 1) / num_threads_x + 1, (num_cols - 1) / num_threads_y + 1);

  InitValuesImpl<<<num_blocks, num_threads>>>(*values, grid_x, grid_y, num_rows, num_cols,
                                              proc_bounds.is_up, proc_bounds.is_low, proc_bounds.is_left, proc_bounds.is_right);
}

__global__ void InitValuesImpl(double* dst, double* grid_x, double* grid_y, size_t num_rows, size_t num_cols,
                               bool is_up, bool is_low, bool is_left, bool is_right) {
  int i = (blockIdx.x * blockDim.x) + threadIdx.x;
  int j = (blockIdx.y * blockDim.y) + threadIdx.y;

  if (i < num_rows && j < num_cols) {
    if ((is_up && i == 0) || (is_low && i == (num_rows - 1)) || (is_left && j == 0) || (is_right && j == num_cols - 1)) {
      matrix_set(dst, i, j, bound_func_k(matrix_get(grid_x, i, j, num_cols), matrix_get(grid_y, i, j, num_cols)), num_cols);
    } else {
      matrix_set(dst, i, j, RAND_CONST, num_cols);
    }
  }
}

void InitGrid_CUDA(double** grid_x, double** grid_y, size_t* num_rows, size_t* num_cols,
                   const GridData& grid_data, const ProcBounds& proc_bounds,
                   size_t start_row_idx, size_t end_row_idx, size_t start_col_idx, size_t end_col_idx) {
  start_row_idx -= !proc_bounds.is_up ? 1 : 0;
  end_row_idx += !proc_bounds.is_low ? 1 : 0;
  start_col_idx -= !proc_bounds.is_left ? 1 : 0;
  end_col_idx += !proc_bounds.is_right ? 1 : 0;

  *num_rows = end_row_idx - start_row_idx;
  *num_cols = end_col_idx - start_col_idx;

  int num_elems = (*num_rows) * (*num_cols);
  CheckCuda(hipMalloc(grid_x, num_elems * sizeof(double)));
  CheckCuda(hipMalloc(grid_y, num_elems * sizeof(double)));

  hipDeviceProp_t devProp;
  CheckCuda(hipGetDeviceProperties(&devProp, 0));

  double* grid_x_cuda = new double[num_elems];
  double* grid_y_cuda = new double[num_elems];

  for (size_t i = 0; i < grid_data.r_num_points; ++i) {
    if (i >= start_row_idx && i < end_row_idx) {
      for (size_t j = 0; j < grid_data.c_num_points; ++j) {
        if (j >= start_col_idx && j < end_col_idx) {
          double cur_r_value = grid_data.r_upper_bound * step_func_k(static_cast<double>(i) / (grid_data.r_num_points - 1), grid_data.q) +
            grid_data.r_lower_bound * (1 - step_func_k(static_cast<double>(i) / (grid_data.r_num_points - 1), grid_data.q));
          double cur_c_value = grid_data.c_upper_bound * step_func_k(static_cast<double>(j) / (grid_data.c_num_points - 1), grid_data.q) +
            grid_data.c_lower_bound * (1 - step_func_k(static_cast<double>(j) / (grid_data.c_num_points - 1), grid_data.q));
          grid_x_cuda[(i - start_row_idx) * (*num_cols) + (j - start_col_idx)] = cur_r_value;
          grid_y_cuda[(i - start_row_idx) * (*num_cols) + (j - start_col_idx)] = cur_c_value;
        }
      }
    }
  }

  CheckCuda(hipMemcpy(*grid_x, grid_x_cuda, num_elems * sizeof(double), hipMemcpyHostToDevice));
  CheckCuda(hipMemcpy(*grid_y, grid_y_cuda, num_elems * sizeof(double), hipMemcpyHostToDevice));
  delete[] grid_x_cuda;
  delete[] grid_y_cuda;
}

void InitMatrix_CUDA(double** matrix, size_t num_rows, size_t num_cols) {
  int num_elems = num_rows * num_cols;
  CheckCuda(hipMalloc(matrix, num_elems * sizeof(double)));

  hipDeviceProp_t devProp;
  CheckCuda(hipGetDeviceProperties(&devProp, 0));

  int temp = (int)(devProp.maxThreadsPerBlock / 2);
  int num_threads = (temp > num_elems) ? num_elems : temp;
  int num_blocks = (num_elems - 1) / num_threads + 1;

  InitMatrixImpl<<<num_blocks, num_threads>>>(*matrix, num_elems);
}

__global__ void InitMatrixImpl(double* dst, size_t num_elems) {
  int i = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (i < num_elems) {
    dst[i] = 0.0;
  }
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void ExchangeRowsCols_CUDA(double* matrix, size_t num_rows, size_t num_cols, const ProcBounds& proc_bounds,
                           std::pair<bool, bool> first_send, std::pair<int, int> left_right_proc, size_t proc_rank) {
  if (proc_bounds.is_up && proc_bounds.is_low && proc_bounds.is_left && proc_bounds.is_right) {
    return;
  }

  // exchange rows
  std::vector<double> data_send(num_cols, 0.0);
  std::vector<double> data_recv(num_cols, 0.0);

  if (!proc_bounds.is_low) {    
    CheckCuda(hipMemcpy(&(data_send[0]), &(matrix[num_cols * (num_rows - 2)]), num_cols * sizeof(double), hipMemcpyDeviceToHost));
    send_receive_vector(data_send,
                        &data_recv,
                        proc_rank + 1,
                        proc_rank + 1,
                        proc_rank,
                        proc_rank + 1);
    CheckCuda(hipMemcpy(&(matrix[num_cols * (num_rows - 1)]), &(data_recv[0]), num_cols * sizeof(double), hipMemcpyHostToDevice));
  }

  if (!proc_bounds.is_up) {
    CheckCuda(hipMemcpy(&(data_send[0]), &(matrix[num_cols]), num_cols * sizeof(double), hipMemcpyDeviceToHost));
    send_receive_vector(data_send,
                        &data_recv,
                        proc_rank - 1,
                        proc_rank - 1,
                        proc_rank,
                        proc_rank - 1);
    CheckCuda(hipMemcpy(&(matrix[0]), &(data_recv[0]), num_cols * sizeof(double), hipMemcpyHostToDevice)); 
  }

  // exchange cols
  data_send.assign(num_rows, 0.0);
  data_recv.assign(num_rows, 0.0);

  if (!proc_bounds.is_left) {
    hipDeviceProp_t devProp;
    CheckCuda(hipGetDeviceProperties(&devProp, 0));
    int num_threads = (int)(devProp.maxThreadsPerBlock / 2);
    int num_blocks = (num_rows * num_cols - 1) / num_threads + 1;

    double* temp;
    CheckCuda(hipMalloc((void**)(&temp), num_rows * sizeof(double)));
    ExtractCol_CUDA<<<num_blocks, num_threads>>>(temp, matrix, 1, num_rows, num_cols);
    CheckCuda(hipMemcpy(&(data_send[0]), temp, num_rows * sizeof(double), hipMemcpyDeviceToHost));

    send_receive_vector(data_send,
                        &data_recv,
                        left_right_proc.first,
                        left_right_proc.first,
                        proc_rank,
                        left_right_proc.first);

    CheckCuda(hipMemcpy(temp, &(data_recv[0]), num_rows * sizeof(double), hipMemcpyHostToDevice));
    InsertCol_CUDA<<<num_blocks, num_threads>>>(matrix, temp, 0, num_rows, num_cols);
    CheckCuda(hipFree(temp));
  }

  if (!proc_bounds.is_right) {
    hipDeviceProp_t devProp;
    CheckCuda(hipGetDeviceProperties(&devProp, 0));
    int num_threads = (int)(devProp.maxThreadsPerBlock / 2);
    int num_blocks = (num_rows * num_cols - 1) / num_threads + 1;

    double* temp;
    CheckCuda(hipMalloc((void**)(&temp), num_rows * sizeof(double)));
    ExtractCol_CUDA<<<num_blocks, num_threads>>>(temp, matrix, num_cols - 2, num_rows, num_cols);
    CheckCuda(hipMemcpy(&(data_send[0]), temp, num_rows * sizeof(double), hipMemcpyDeviceToHost));

    send_receive_vector(data_send,
                        &data_recv,
                        left_right_proc.second,
                        left_right_proc.second,
                        proc_rank,
                        left_right_proc.second);

    CheckCuda(hipMemcpy(temp, &(data_recv[0]), num_rows * sizeof(double), hipMemcpyHostToDevice));
    InsertCol_CUDA<<<num_blocks, num_threads>>>(matrix, temp, num_cols - 1, num_rows, num_cols);
    CheckCuda(hipFree(temp));
  }
}

__global__ void ExtractCol_CUDA(double* dst, double* src, size_t col, size_t num_rows, size_t num_cols) {
  int i = (blockIdx.x * blockDim.x) + threadIdx.x;
  
  if (i < num_rows * num_cols) {
    if (i % num_cols == col) {
      dst[i / num_cols] = src[i];
    }
  }
}

__global__ void InsertCol_CUDA(double* dst, double* src, size_t col, size_t num_rows, size_t num_cols) {
  int i = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (i < num_rows * num_cols) {
    if (i % num_cols == col) {
      dst[i] = src[i / num_cols];
    }
  }
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void CountResiduals_CUDA(double* residuals, double* values_laplass, double* grid_x, double* grid_y,
                         size_t num_rows, size_t num_cols, const ProcBounds& proc_bounds) {
  hipDeviceProp_t devProp;
  CheckCuda(hipGetDeviceProperties(&devProp, 0));

  int temp = (int)(sqrt(devProp.maxThreadsPerBlock) / 2);
  int num_threads_x = (temp > num_rows) ? num_rows : temp;
  int num_threads_y = (temp > num_cols) ? num_cols : temp;
  dim3 num_threads(num_threads_x, num_threads_y);
  dim3 num_blocks((num_rows - 1) / num_threads_x + 1, (num_cols - 1) / num_threads_y + 1);

  CountResidualsImpl<<<num_blocks, num_threads>>>(residuals, values_laplass, grid_x, grid_y, num_rows, num_cols,
                                                  proc_bounds.is_up, proc_bounds.is_low, proc_bounds.is_left, proc_bounds.is_right);
}

__global__ void CountResidualsImpl(double* residuals, double* values_laplass, double* grid_x, double* grid_y,
                                   size_t num_rows, size_t num_cols, bool is_up, bool is_low, bool is_left, bool is_right) {
  int i = (blockIdx.x * blockDim.x) + threadIdx.x;
  int j = (blockIdx.y * blockDim.y) + threadIdx.y;

  if (i < num_rows && j < num_cols) {
    if ((j == 0 && is_left) || (j == (num_cols - 1) && is_right) || (i == 0 && is_up) || (i == (num_rows - 1) && is_low)) {
      matrix_set(residuals, i, j, 0.0, num_cols);
    } else {
      double r = matrix_get(grid_x, i, j, num_cols);
      double c = matrix_get(grid_y, i, j, num_cols);
      matrix_set(residuals, i, j, matrix_get(values_laplass, i, j, num_cols) - main_func_k(r, c), num_cols);
    }
  }
}

double CountPreError_CUDA(double* values, double* temp_matrix, double* grid_x, double* grid_y, size_t num_rows, size_t num_cols) {
  hipDeviceProp_t devProp;
  CheckCuda(hipGetDeviceProperties(&devProp, 0));

  int temp = (int)(sqrt(devProp.maxThreadsPerBlock) / 2);
  int num_threads_x = (temp > num_rows) ? num_rows : temp;
  int num_threads_y = (temp > num_cols) ? num_cols : temp;
  dim3 num_threads(num_threads_x, num_threads_y);
  dim3 num_blocks((num_rows - 1) / num_threads_x + 1, (num_cols - 1) / num_threads_y + 1);

  CountPreErrorImpl<<<num_blocks, num_threads>>>(temp_matrix, values, grid_x, grid_y, num_rows, num_cols);

  return ProductByPointAndSum_CUDA(temp_matrix, temp_matrix, grid_x, grid_y, num_rows, num_cols);
}

__global__ void CountPreErrorImpl(double* psi, double* values, double* grid_x, double* grid_y, size_t num_rows, size_t num_cols) {
  int i = (blockIdx.x * blockDim.x) + threadIdx.x;
  int j = (blockIdx.y * blockDim.y) + threadIdx.y;

  if (i < num_rows && j < num_cols) {
    double r = matrix_get(grid_x, i, j, num_cols);
    double c = matrix_get(grid_y, i, j, num_cols);
    matrix_set(psi, i, j, matrix_get(values, i, j, num_cols) - true_func_k(r, c), num_cols);
  }
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void CountNewValues_CUDA(double* values, double* old_values, double* gradients,
                         double* temp_matrix, double tau, size_t num_rows, size_t num_cols) {
  CopyMatrix_CUDA(old_values, values, num_rows, num_cols);
  ProdMatrixByScalar_CUDA(temp_matrix, gradients, tau, num_rows, num_cols);
  DiffMatrices_CUDA(values, old_values, temp_matrix, num_rows, num_cols);
}

double CountValuesDifference_CUDA(double* values, double* old_values, double* temp_matrix,
                                  double* grid_x, double* grid_y, size_t num_rows, size_t num_cols) {
  DiffMatrices_CUDA(temp_matrix, values, old_values, num_rows, num_cols);
  return ProductByPointAndSum_CUDA(temp_matrix, temp_matrix, grid_x, grid_y, num_rows, num_cols);
}

void CountGradients_CUDA(double* gradients, double* residuals, double* temp_matrix,
                         double alpha, size_t num_rows, size_t num_cols, bool first_iter) {
  if (first_iter) {
    CopyMatrix_CUDA(gradients, residuals, num_rows, num_cols);
  } else {
    ProdMatrixByScalar_CUDA(temp_matrix, gradients, alpha, num_rows, num_cols);
    DiffMatrices_CUDA(gradients, residuals, temp_matrix, num_rows, num_cols);
  }
}

} // namespace DTS
