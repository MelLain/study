#include "hip/hip_runtime.h"
#include "common.h"

namespace DTS {

__device__ double main_func_k(double r, double c) {
  return 8 - 12 * pow(r, 2) - 12 * pow(c, 2);
}

__device__ double bound_func_k(double r, double c) {
  return pow((1 - pow(r, 2)), 2) + pow((1 - pow(c, 2)), 2);
}

__device__ double true_func_k(double r, double c) {
  return pow((1 - pow(r, 2)), 2) + pow((1 - pow(c, 2)), 2);
}

__host__ __device__ double step_func_k(double value, double q) {
  return (pow(1 + value, q) - 1) / (pow(2.0, q) - 1);
}


__device__ double matrix_get(double* data, size_t row_index, size_t col_index, size_t num_cols) {
  return data[num_cols * row_index + col_index];
}

__device__ void matrix_set(double* data, size_t row_index, size_t col_index, double value, size_t num_cols) {
  data[num_cols * row_index + col_index] = value;
}

__device__ double grid_r_step(double* data_x, double* data_y, size_t row, size_t num_cols) {
  return row > 0 ? matrix_get(data_x, row, 0, num_cols) - matrix_get(data_x, row - 1, 0, num_cols) : 0.0;
}

__device__ double grid_c_step(double* data_x, double* data_y, size_t col, size_t num_cols) {
  return col > 0 ? matrix_get(data_y, 0, col, num_cols) - matrix_get(data_y, 0, col - 1, num_cols) : 0.0;
}

} // namespace DTS
