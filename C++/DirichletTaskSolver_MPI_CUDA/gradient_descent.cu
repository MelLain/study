#include "hip/hip_runtime.h"
#include <cmath>
#include <string>
#include <utility>

#include "mpi_helpers.h"
#include "gradient_descent.h"

namespace DTS {

GradientDescent::GradientDescent(const GridData& grid_data, const ProcBounds& proc_bounds,
                                 size_t num_procs, size_t proc_rank, size_t num_row_procs, size_t num_points,
                                 size_t start_row_idx, size_t end_row_idx, size_t start_col_idx, size_t end_col_idx)
  : proc_bounds_(proc_bounds)
  , num_procs_(num_procs)
  , proc_rank_(proc_rank)
  , num_points_(num_points)
  , first_send_(std::make_pair((proc_rank % num_row_procs) % 2 == 1, (proc_rank / num_row_procs) % 2 == 0))
  , left_right_proc_(std::make_pair(proc_bounds.is_left ? -1 : proc_rank - num_row_procs,
                                    proc_bounds.is_right ? -1 : proc_rank + num_row_procs))
  , num_rows_(0)
  , num_cols_(0)
{
  CheckCuda(hipSetDevice(0));

  InitGrid_CUDA(&grid_x_cuda_, &grid_y_cuda_, &num_rows_, &num_cols_, grid_data, proc_bounds,
                start_row_idx, end_row_idx, start_col_idx, end_col_idx);
  InitValues_CUDA(&values_cuda_, grid_x_cuda_, grid_y_cuda_, num_rows_, num_cols_, proc_bounds);

  InitMatrix_CUDA(&values_laplass_cuda_, num_rows_, num_cols_);
  InitMatrix_CUDA(&old_values_cuda_, num_rows_, num_cols_);
  InitMatrix_CUDA(&residuals_cuda_, num_rows_, num_cols_);
  InitMatrix_CUDA(&residuals_laplass_cuda_, num_rows_, num_cols_);
  InitMatrix_CUDA(&gradients_cuda_, num_rows_, num_cols_);
  InitMatrix_CUDA(&gradients_laplass_cuda_, num_rows_, num_cols_);
  InitMatrix_CUDA(&temp_matrix_cuda_, num_rows_, num_cols_);
}

std::pair<size_t, double> GradientDescent::FitModel() {
  size_t processed_iter = 0;
  double error = 1e+10;
  while (true) {
    FlagType flag;
    if (proc_rank_ == 0) {
      send_flag_to_all(num_procs_, START_ITER);
    } else {
      receive_flag(&flag, 0, proc_rank_);
      if (flag == TERMINATE) {
        double pre_err = CountPreError_CUDA(values_cuda_, temp_matrix_cuda_,
                                            grid_x_cuda_, grid_y_cuda_, num_rows_, num_cols_);
        send_value(pre_err, 0, proc_rank_);
        break;
      }
    }

    ExchangeRowsCols_CUDA(values_cuda_, num_rows_, num_cols_, proc_bounds_, first_send_, left_right_proc_, proc_rank_);

    // step 1: count residuals
    FivePointsLaplass_CUDA(values_laplass_cuda_, values_cuda_, grid_x_cuda_, grid_y_cuda_, num_rows_, num_cols_);
    ExchangeRowsCols_CUDA(values_laplass_cuda_, num_rows_, num_cols_, proc_bounds_, first_send_, left_right_proc_, proc_rank_);

    CountResiduals_CUDA(residuals_cuda_, values_laplass_cuda_, grid_x_cuda_, grid_y_cuda_, num_rows_, num_cols_, proc_bounds_);

    FivePointsLaplass_CUDA(residuals_laplass_cuda_, residuals_cuda_, grid_x_cuda_, grid_y_cuda_, num_rows_, num_cols_);
    ExchangeRowsCols_CUDA(residuals_laplass_cuda_, num_rows_, num_cols_, proc_bounds_, first_send_, left_right_proc_, proc_rank_);

    // step 2: count alpha
    double alpha_den = ProductByPointAndSum_CUDA(gradients_laplass_cuda_, gradients_cuda_,
                                                 grid_x_cuda_, grid_y_cuda_, num_rows_, num_cols_);
    double alpha_nom = ProductByPointAndSum_CUDA(residuals_laplass_cuda_, gradients_cuda_,
                                                 grid_x_cuda_, grid_y_cuda_, num_rows_, num_cols_);

    if (proc_rank_ == 0) {
      alpha_den = collect_value_from_all(num_procs_) + alpha_den;
      send_value_to_all(num_procs_, alpha_den);
      alpha_nom = collect_value_from_all(num_procs_) + alpha_nom;
      send_value_to_all(num_procs_, alpha_nom);
    } else {
      send_value(alpha_den, 0, proc_rank_);
      receive_value(&alpha_den, 0, proc_rank_);
      send_value(alpha_nom, 0, proc_rank_);
      receive_value(&alpha_nom, 0, proc_rank_);
    }
    double alpha = alpha_den > 0.0 ? alpha_nom / alpha_den : 0.0;

    // step 3: count new gradients
    CountGradients_CUDA(gradients_cuda_, residuals_cuda_, temp_matrix_cuda_,
                        alpha, num_rows_, num_cols_, processed_iter == 0); 
    processed_iter++;

    // step 4: count new gradients laplass
    FivePointsLaplass_CUDA(gradients_laplass_cuda_, gradients_cuda_,
                           grid_x_cuda_, grid_y_cuda_, num_rows_, num_cols_);

    // step 5: count new tau
    double tau_den = ProductByPointAndSum_CUDA(gradients_laplass_cuda_, gradients_cuda_,
                                               grid_x_cuda_, grid_y_cuda_, num_rows_, num_cols_);
    double tau_nom = ProductByPointAndSum_CUDA(residuals_cuda_, gradients_cuda_,
                                               grid_x_cuda_, grid_y_cuda_, num_rows_, num_cols_);
    if (proc_rank_ == 0) {
      tau_den = collect_value_from_all(num_procs_) + tau_den;
      send_value_to_all(num_procs_, tau_den);
      tau_nom = collect_value_from_all(num_procs_) + tau_nom;
      send_value_to_all(num_procs_, tau_nom);
    } else {
      send_value(tau_den, 0, proc_rank_);
      receive_value(&tau_den, 0, proc_rank_);
      send_value(tau_nom, 0, proc_rank_);
      receive_value(&tau_nom, 0, proc_rank_);
    }
    double tau = tau_den > 0.0 ? tau_nom / tau_den : 0.0;

    // step 6: count new values and process error
    CountNewValues_CUDA(values_cuda_, old_values_cuda_, gradients_cuda_, temp_matrix_cuda_, tau, num_rows_, num_cols_);
    double val_diff = CountValuesDifference_CUDA(values_cuda_, old_values_cuda_, temp_matrix_cuda_,
                                            grid_x_cuda_, grid_y_cuda_, num_rows_, num_cols_);
    if (proc_rank_ == 0) {
      double difference = sqrt(collect_value_from_all(num_procs_) + val_diff);
      if (difference < EPS) {
        send_flag_to_all(num_procs_, TERMINATE);
        double pre_error = CountPreError_CUDA(values_cuda_, temp_matrix_cuda_, grid_x_cuda_,
                                               grid_y_cuda_, num_rows_, num_cols_);
        error = sqrt(collect_value_from_all(num_procs_) + pre_error);
        break;
      }      
    } else {
      send_value(val_diff, 0, proc_rank_);
    }
  }
  return std::make_pair(processed_iter, error);
}

/*
void GradientDescent::save_results_file() {
  std::ofstream out_value_file;
  std::ofstream out_true_file;

  size_t start_row_shift = proc_bounds_.is_up ? 0 : 1;
  size_t end_row_shift = proc_bounds_.is_low ? 0 : 1;
  size_t start_col_shift = proc_bounds_.is_left ? 0 : 1;
  size_t end_col_shift = proc_bounds_.is_right ? 0 : 1;

  out_value_file.open("VALUE_PART_POINTS_" + std::to_string(num_points_) + "_PROC_" + std::to_string(proc_rank_));
  out_true_file.open("TRUE_PART_POINTS_" + std::to_string(num_points_) + "_PROC_" + std::to_string(proc_rank_));

  for (size_t i = start_row_shift; i < values_->num_rows() - end_row_shift; ++i) {
    for (size_t j = start_col_shift; j < values_->num_cols() - end_col_shift; ++j) {
      out_value_file << (*values_)(i, j) << ", ";
      out_true_file << true_func((*grid_)(i, j)) << ", ";                                                                                       
    }
    out_value_file << std::endl;
    out_true_file << std::endl;
  }

  out_value_file.close();
  out_true_file.close();
}*/

}  // namespace DTS
